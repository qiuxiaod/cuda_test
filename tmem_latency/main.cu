#include "hip/hip_runtime.h"
#include <iostream>
#include "inline_ptx_func.hpp"
#include <hip/hip_runtime.h>

#define SHARED_MEM_SIZE 32
#define THD_NUM 128
#define WARP_SIZE 32

#ifndef REP
#define REP 8
#endif

// CUDA kernel to perform memory loads from shared memory and measure latency
__global__ void benchmarkTMEMLoadLatency(unsigned long long *d_start, unsigned long long *d_end, uint32_t* data) {
    // Declare shared memory
    __shared__ uint32_t sharedMem[SHARED_MEM_SIZE];
    int tid = threadIdx.x;
    int warp_id = tid / WARP_SIZE;
    if(warp_id == 0){
        tmem_allocate(sharedMem, 512);
    }
    __syncthreads();

    uint32_t tmem_ptr = sharedMem[0];
    
    uint32_t val_array[REP];

    #pragma unroll
    for(int i = 0; i < REP; i++){
        val_array[i] = data[tid + i * THD_NUM];
    }
    
    tmem_st_32dp32bNx<REP>(tmem_ptr, val_array);

    fence_view_async_tmem_store();

    __syncthreads();
    unsigned long long start, end;

    uint32_t val_array_tmp[REP];
    __syncwarp();
    start = clock64();

    tmem_ld_32dp32bNx<REP>(tmem_ptr, val_array_tmp);
    fence_view_async_tmem_load();

    val_array[0] += val_array_tmp[0];

    end = clock64();

    __syncthreads();

    if(warp_id == 0){
        tmem_free(tmem_ptr, 512);
    }
    __syncthreads();

    // Write the start and end times to global memory
    if (tid == 0) {
        d_start[0] = start;
        d_end[0] = end;
    }

    #pragma unroll
    for(int i = 0; i < REP; i++){
        data[tid + i * THD_NUM] = val_array[i];
    }
}

int main() {
    // Allocate memory on the device for start and end times
    unsigned long long *d_start, *d_end;
    uint32_t *d_data;
    hipMalloc(&d_data, sizeof(uint32_t) * THD_NUM * REP);
    hipMalloc(&d_start, sizeof(unsigned long long));
    hipMalloc(&d_end, sizeof(unsigned long long));

    // Launch the kernel
    benchmarkTMEMLoadLatency<<<1, THD_NUM>>>(d_start, d_end, d_data);

    // Copy the start and end times back to the host
    unsigned long long h_start, h_end;
    hipMemcpy(&h_start, d_start, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipMemcpy(&h_end, d_end, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Calculate the latency in clock cycles
    double latency = (h_end - h_start);

    // Print the result
    std::cout << "TMEM Load Latency: " << latency << " clock cycles" << std::endl;

    // Clean up
    hipFree(d_start);
    hipFree(d_end);

    return 0;
}