#include "hip/hip_runtime.h"
#include <iostream>
#include "inline_ptx_func.hpp"
#include <hip/hip_runtime.h>

#define SHARED_MEM_SIZE 32
#define THD_NUM 128
#define WARP_SIZE 32

#ifndef REP
#define REP 8
#endif

// TEST_MODE == 0 means load only
// TEST_MODE == 1 means store + load serialized
#ifndef TEST_MODE
#define TEST_MODE 0
#endif

// CUDA kernel to perform memory loads from shared memory and measure latency
__global__ void benchmarkTMEMLoadLatency(unsigned long long *d_start, unsigned long long *d_end, uint32_t* data) {
    // Declare shared memory
    __shared__ uint32_t sharedMem[SHARED_MEM_SIZE];
    int tid = threadIdx.x;
    int warp_id = tid / WARP_SIZE;
    if(warp_id == 0){
        tmem_allocate(sharedMem, 512);
    }
    __syncthreads();

    uint32_t tmem_ptr = sharedMem[0];
    uint32_t tmem_ptr1 = tmem_ptr + 128;
    
    uint32_t val_array[REP];

    #pragma unroll
    for(int i = 0; i < REP; i++){
        val_array[i] = data[tid + i * THD_NUM];
    }
    
    tmem_st_32dp32bNx<REP>(tmem_ptr, val_array);
    tmem_st_32dp32bNx<REP>(tmem_ptr1, val_array);

    fence_view_async_tmem_store();

    __syncthreads();
    unsigned long long start[2];
    unsigned long long end[2];
    uint32_t val_array_tmp[REP];

#if TEST_MODE == 0
        __syncthreads();
        __syncwarp();
        start[0] = clock64();

        tmem_ld_32dp32bNx<REP>(tmem_ptr, val_array_tmp);
        fence_view_async_tmem_load();

        val_array[0] += val_array_tmp[0];

        end[0] = clock64();
#elif TEST_MODE == 1
        __syncthreads();
        __syncwarp();
        start[0] = clock64();

        tmem_st_32dp32bNx<REP>(tmem_ptr, val_array);
        fence_view_async_tmem_store();
        
        end[0] = clock64();

        tmem_ld_32dp32bNx<REP>(tmem_ptr, val_array_tmp);
        fence_view_async_tmem_load();

        val_array[0] += val_array_tmp[0];

#elif TEST_MODE == 2
        uint32_t val_array_tmp1[REP];
        __syncthreads();
        __syncwarp();
        start[0] = clock64();

        tmem_st_32dp32bNx<REP>(tmem_ptr, val_array);
        tmem_ld_32dp32bNx<REP>(tmem_ptr1, val_array_tmp1);
        fence_view_async_tmem_store();
        tmem_ld_32dp32bNx<REP>(tmem_ptr, val_array_tmp);
        fence_view_async_tmem_load();

        val_array[0] += val_array_tmp[0] + val_array_tmp1[0];
        end[0] = clock64();
#endif
    // if(warp_id < 4){

    // }


    __syncthreads();

    if(warp_id == 0){
        tmem_free(tmem_ptr, 512);
    }
    __syncthreads();

    // Write the start and end times to global memory
    if (tid == 0) {
        d_start[0] = start[0];
        d_end[0] = end[0];
    }

    #pragma unroll
    for(int i = 0; i < REP; i++){
        data[tid + i * THD_NUM] = val_array[i];
    }
}

int main() {
    // Allocate memory on the device for start and end times
    unsigned long long *d_start, *d_end;
    uint32_t *d_data;
    hipMalloc(&d_data, sizeof(uint32_t) * THD_NUM * REP);
    hipMalloc(&d_start, sizeof(unsigned long long) * 2);
    hipMalloc(&d_end, sizeof(unsigned long long) * 2);

    // Launch the kernel
    benchmarkTMEMLoadLatency<<<1, THD_NUM>>>(d_start, d_end, d_data);

    // Copy the start and end times back to the host
    unsigned long long* h_start = (unsigned long long*)malloc(sizeof(unsigned long long) * 2);
    unsigned long long* h_end = (unsigned long long*)malloc(sizeof(unsigned long long) * 2);

    hipMemcpy(h_start, d_start, sizeof(unsigned long long) * 2, hipMemcpyDeviceToHost);
    hipMemcpy(h_end, d_end, sizeof(unsigned long long) * 2, hipMemcpyDeviceToHost);

    #if TEST_MODE == 0
        // Calculate the latency in clock cycles
        double latency = (h_end[0] - h_start[0]);
        // Print the result
        std::cout << "TMEM Load[0] Latency: " << latency << " clock cycles" << std::endl;
    #elif TEST_MODE == 1
        double latency = (h_end[0] - h_start[0]);
        std::cout << "TMEM Store[0] + Load[0] Latency: " << latency << " clock cycles" << std::endl;
    #elif TEST_MODE == 2
        double latency = (h_end[0] - h_start[0]);
        std::cout << "TMEM Store[0] + Load[1] + Load[0] Latency: " << latency << " clock cycles" << std::endl;
    #endif
    // Clean up
    hipFree(d_start);
    hipFree(d_end);
    free(h_start);
    free(h_end);
    hipFree(d_data);

    return 0;
}